#include "hip/hip_runtime.h"
#include <cstring>

#include "array_2d.hpp"
#include "collision_checker.hpp"
#include "cuda_util.hpp"
#include "map.hpp"
#include "obstacle_manager.hpp"

namespace gpu_planning {

CollisionCheckResult::CollisionCheckResult() : result{false} {}

CollisionCheckResult::CollisionCheckResult(bool result, uint8_t obstacle_id)
    : result{result}, obstacle_id{obstacle_id} {}

CollisionChecker::CollisionChecker()
    : check_block_size_{0},
      device_configuration_buf_{},
      device_result_buf_{},
      map_{nullptr},
      robot_{nullptr},
      obstacle_manager_{nullptr},
      log_{nullptr} {}

CollisionChecker::CollisionChecker(DeviceMap* map, DeviceRobot* robot,
                                   ObstacleManager* obstacle_manager,
                                   Logger* log)
    : check_block_size_{100},
      device_configuration_buf_{check_block_size_},
      device_result_buf_{check_block_size_},
      map_{map},
      robot_{robot},
      obstacle_manager_{obstacle_manager},
      log_{log} {}

__global__ void check_collisions(Map* map, Robot* robot,
                                 Array<Configuration>* configurations,
                                 Array<CollisionCheckResult>* results,
                                 size_t num_checks) {
  for (size_t i = threadIdx.x; i < num_checks; i += blockDim.x) {
    const Pose<float> ee = robot->fk_ee((*configurations)[i]);
    const Cell& cell = map->get(ee.position);

    (*results)[i] = CollisionCheckResult(cell.value >= 1.f, cell.id);
  }
}

void CollisionChecker::check(const std::vector<Configuration>& configurations) {
  LOG_DEBUG(log_) << "Checking " << configurations.size()
                  << " configurations for collisions in blocks of "
                  << check_block_size_;
  std::vector<CollisionCheckResult> result;
  result.resize(configurations.size());

  size_t num_iterations = (configurations.size() - 1) / check_block_size_ + 1;
  for (size_t i = 0; i < num_iterations; ++i) {
    size_t block_remaining =
        min(check_block_size_, configurations.size() - i * check_block_size_);

    device_configuration_buf_.memcpy_set(configurations, i * check_block_size_,
                                         block_remaining);
    check_collisions<<<1, 32>>>(map_->device_map(), robot_->device_handle(),
                                device_configuration_buf_.device_handle(),
                                device_result_buf_.device_handle(),
                                block_remaining);
    device_result_buf_.memcpy_get(result, i * check_block_size_,
                                  block_remaining);
  }

  for (size_t i = 0; i < result.size(); ++i) {
    if (result[i].result) {
      const std::string obst_name =
          obstacle_manager_->get_obstacle_name(result[i].obstacle_id);
      LOG_DEBUG(log_) << "Configuration " << i << ": X   (" << obst_name << ")";
    } else {
      LOG_DEBUG(log_) << "Configuration " << i << ":   X";
    }
  }
}

}  // namespace gpu_planning
