#include "map.hpp"

namespace gpu_planning {

Map::Map() : map_{nullptr} {}

Map::Map(size_t width, size_t height) : map_{new hipPitchedPtr()} {
  hipExtent extent = make_hipExtent(width, height, sizeof(float));
  if (hipMalloc3D(map_.get(), extent) != hipSuccess) {
    throw std::runtime_error{"Could not allocate map memory"};
  }

  if (hipMemset3D(*map_.get(), 0, extent) != hipSuccess) {
    throw std::runtime_error{"Could not clear map memory"};
  }
}

Map::~Map() {
  if (map_ != nullptr) {
    hipFree(map_->ptr);
  }
}

}  // namespace gpu_planning
