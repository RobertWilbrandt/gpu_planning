#include <iostream>

#include "map.hpp"

#define CHECK_CUDA(fun, mes)                                              \
  hipError_t err = fun;                                                  \
  if (err != hipSuccess) {                                               \
    throw std::runtime_error{std::string(mes) + hipGetErrorString(err)}; \
  }

namespace gpu_planning {

DeviceArray2D::DeviceArray2D() : extent_{nullptr}, pitched_ptr_{nullptr} {}

DeviceArray2D::DeviceArray2D(size_t width, size_t height, size_t mem_size)
    : extent_{nullptr}, pitched_ptr_{nullptr} {
  extent_ = new hipExtent(make_hipExtent(width * mem_size, height, mem_size));
  pitched_ptr_ = new hipPitchedPtr();

  hipExtent* extent = (hipExtent*)extent_;
  hipPitchedPtr* pitched_ptr = (hipPitchedPtr*)pitched_ptr_;

  CHECK_CUDA(hipMalloc3D(pitched_ptr, *extent),
             "Could not allocate 2D device array: ");
}

DeviceArray2D::~DeviceArray2D() {
  if (extent_ != nullptr) {
    free(extent_);
  }
  if (pitched_ptr_ != nullptr) {
    hipFree(((hipPitchedPtr*)pitched_ptr_)->ptr);
    free(pitched_ptr_);
  }
}

size_t DeviceArray2D::width() const {
  hipExtent* extent = (hipExtent*)extent_;
  return extent->width / extent->depth;
}

size_t DeviceArray2D::height() const {
  hipExtent* extent = (hipExtent*)extent_;
  return extent->height;
}

void DeviceArray2D::clear() {
  hipExtent* extent = (hipExtent*)extent_;
  hipPitchedPtr* pitched_ptr = (hipPitchedPtr*)pitched_ptr_;

  CHECK_CUDA(hipMemset3D(*pitched_ptr, 0, *extent),
             "Could not clear 2D device array: ");
}

void DeviceArray2D::read(size_t x, size_t y, size_t w, size_t h, void* dest) {
  hipExtent* extent = (hipExtent*)extent_;
  hipPitchedPtr* pitched_ptr = (hipPitchedPtr*)pitched_ptr_;

  size_t dpitch = w * extent->depth;
  void* src = (unsigned char*)pitched_ptr->ptr + x * extent->depth +
              y * pitched_ptr->pitch;
  size_t spitch = pitched_ptr->pitch;
  size_t width = w * extent->depth;
  size_t height = h;

  CHECK_CUDA(hipMemcpy2D(dest, dpitch, src, spitch, width, height,
                          hipMemcpyDeviceToHost),
             "Could not read array from device: ");
}

void DeviceArray2D::write(size_t x, size_t y, size_t w, size_t h, void* src) {
  hipExtent* extent = (hipExtent*)extent_;
  hipPitchedPtr* pitched_ptr = (hipPitchedPtr*)pitched_ptr_;

  void* dst = (unsigned char*)pitched_ptr->ptr + x * extent->depth +
              y * pitched_ptr->pitch;
  size_t dpitch = pitched_ptr->pitch;
  size_t spitch = w * extent->depth;
  size_t width = w * extent->depth;
  size_t height = h;

  CHECK_CUDA(hipMemcpy2D(dst, dpitch, src, spitch, width, height,
                          hipMemcpyHostToDevice),
             "Could not write data to device array: ");
}

Map::Map() : map_() {}

Map::Map(size_t width, size_t height) : map_{width, height, sizeof(float)} {}

Map::~Map() {}

}  // namespace gpu_planning
