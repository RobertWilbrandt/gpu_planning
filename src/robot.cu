#include "hip/hip_runtime.h"
#include <math.h>

#include <stdexcept>

#include "cuda_util.hpp"
#include "device_robot.cuh"
#include "robot.hpp"

namespace gpu_planning {

Configuration::Configuration() : joints{0.f, 0.f, 0.f} {}

Configuration::Configuration(float j1, float j2, float j3)
    : joints{j1, j2, j3} {}

Robot::Robot()
    : device_robot_{nullptr},
      base_{0.f, 0.f},
      l1_{0.f},
      l2_{0.f},
      ee_w_{0.f},
      ee_h_{0.f} {}

Robot::Robot(Point base, float l1, float l2, float ee_w, float ee_h)
    : device_robot_{nullptr},
      base_{base},
      l1_{l1},
      l2_{l2},
      ee_w_{ee_w},
      ee_h_{ee_h} {
  CHECK_CUDA(hipMalloc(&device_robot_, sizeof(DeviceRobot)),
             "Could not allocate device storage for robot description");

  DeviceRobot device_robot(base_.x, base_.y, l1, l2, ee_w, ee_h);
  CHECK_CUDA(hipMemcpy(device_robot_, &device_robot, sizeof(DeviceRobot),
                        hipMemcpyHostToDevice),
             "Could not memcpy robot description to device");
}

Robot::~Robot() {
  if (device_robot_ != nullptr) {
    CHECK_CUDA(hipFree(device_robot_),
               "Could not free device robot description");
  }
}

Point Robot::base() const { return base_; }

Point Robot::fk_elbow(const Configuration& conf) const {
  return base_ + Point(l1_ * sin(conf.joints[0]), l1_ * cos(conf.joints[0]));
}

Point Robot::fk_ee(const Configuration& conf) const {
  float comp_a = conf.joints[0] + conf.joints[1];
  return fk_elbow(conf) + Point(l2_ * sin(comp_a), l2_ * cos(comp_a));
}

DeviceRobot* Robot::device_robot() const { return device_robot_; }

}  // namespace gpu_planning
