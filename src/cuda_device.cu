#include "hip/hip_runtime.h"
#include "cuda_device.hpp"

void check_cuda(hipError_t err) {
  if (err != hipSuccess) {
    throw std::runtime_error(std::string("CUDA error: ") +
                             hipGetErrorString(err));
  }
}

namespace gpu_planning {

void cuda_set_device(int dev, logger& log) {
  check_cuda(hipSetDevice(dev));
  LOG_INFO(log) << "Using CUDA device " << dev;
}

void cuda_list_devices(logger& log) {
  int device_count;
  check_cuda(hipGetDeviceCount(&device_count));

  int used_device;
  check_cuda(hipGetDevice(&used_device));

  LOG_INFO(log) << "Found " << device_count << " CUDA devices:";
  for (int i = 0; i < device_count; ++i) {
    hipDeviceProp_t prop;
    check_cuda(hipGetDeviceProperties(&prop, i));

    std::string selected_string;
    if (i == used_device) {
      selected_string = "*";
    } else {
      selected_string = " ";
    }

    LOG_INFO(log) << "- " << selected_string << i << ": " << prop.name
                  << ", PCI " << prop.pciDomainID << ':' << prop.pciBusID << ':'
                  << prop.pciDeviceID << ", SM " << prop.major << '.'
                  << prop.minor;
  }
}

}  // namespace gpu_planning
