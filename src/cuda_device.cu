#include "hip/hip_runtime.h"
#include "cuda_device.hpp"

void check_cuda(hipError_t err) {
  if (err != hipSuccess) {
    throw std::runtime_error(std::string("CUDA error: ") +
                             hipGetErrorString(err));
  }
}

namespace gpu_planning {

void cuda_list_devices(logger& log) {
  int device_count;
  check_cuda(hipGetDeviceCount(&device_count));

  LOG_INFO(log) << "Found " << device_count << " CUDA devices:";
  for (int i = 0; i < device_count; ++i) {
    hipDeviceProp_t prop;
    check_cuda(hipGetDeviceProperties(&prop, i));

    LOG_INFO(log) << "- " << i << ": " << prop.name << ", PCI "
                  << prop.pciDomainID << ':' << prop.pciBusID << ':'
                  << prop.pciDeviceID << ", SM " << prop.major << '.'
                  << prop.minor;
  }
}

}  // namespace gpu_planning
